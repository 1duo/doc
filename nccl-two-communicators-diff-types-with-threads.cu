#include <stdio.h>
#include "hip/hip_runtime.h"
#include "nccl.h"
#include "mpi.h"
#include <unistd.h>
#include <stdint.h>
#include <iostream>
#include <string>
#include <thread>
#include <chrono>

#define MPI_CHECK(cmd) do {			\
    int e = cmd;				\
    if (e != MPI_SUCCESS) {			\
      printf("Failed: MPI error %s:%d '%d'\n",	\
	     __FILE__,__LINE__, e);		\
      exit(EXIT_FAILURE);			\
    }						\
  } while(0)

#define CUDA_CHECK(cmd) do {				\
    hipError_t e = cmd;				\
    if (e != hipSuccess) {				\
      printf("Failed: Cuda error %s:%d '%s'\n",		\
	     __FILE__,__LINE__,hipGetErrorString(e));	\
      exit(EXIT_FAILURE);				\
    }							\
  } while(0)

#define NCCL_CHECK(cmd) do {				\
    ncclResult_t r = cmd;				\
    if (r != ncclSuccess) {				\
      printf("Failed, NCCL error %s:%d '%s'\n",		\
	     __FILE__,__LINE__,ncclGetErrorString(r));	\
      exit(EXIT_FAILURE);				\
    }							\
  } while(0)

static uint64_t getHostHash(const char* string) {
  // based on DJB2, result = result * 33 + char
  uint64_t result = 5381;
  for (int c = 0; string[c] != '\0'; c++) {
    result = ((result << 5) + result) + string[c];
  }
  return result;
}

static void getHostName(char* hostname, int maxlen) {
  gethostname(hostname, maxlen);
  for (int i = 0; i < maxlen; i++) {
    if (hostname[i] == '.') {
      hostname[i] = '\0';
      return;
    }
  }
}

void runAllReduce(int myRank, int typeId, size_t count, void* sendbuff, void* recvbuff, ncclComm_t comm, hipStream_t stream) {
  const std::string dataType = typeId == 0 ? "float16" : "float32";
  std::cout << dataType << " Rank " << myRank << " before ncclAllReduce" << std::endl;
  switch (typeId) {
  case 0:
    NCCL_CHECK(ncclAllReduce((const void*)sendbuff, (void*)recvbuff, count,
			     ncclHalf, ncclSum, comm, stream));
    break;
  case 1:
    NCCL_CHECK(ncclAllReduce((const void*)sendbuff, (void*)recvbuff, count,
			     ncclFloat, ncclSum, comm, stream));
    break;
  default:
    std::cout << "Wrong data type" << std::endl;
  }
  std::cout << dataType << " Rank " << myRank << " before stream sync" << std::endl;
  CUDA_CHECK(hipStreamSynchronize(stream));
  std::cout << dataType << " Rank " << myRank << " finish stream sync" << std::endl;
}

void delayRunAllReduce(int myRank, int typeId, size_t count, void* sendbuff, void* recvbuff, ncclComm_t comm, hipStream_t stream) {
  for (int i = 0; i < 5 + typeId; ++i) {
    // std::cout << "Sleep [" << i << "] ..." << std::endl;
    std::this_thread::sleep_for(std::chrono::seconds(1));
  }
  runAllReduce(myRank, typeId, count, sendbuff, recvbuff, comm, stream);
}

int main(int argc, char* argv[]) {

  int size = 256 * 1024 * 1024; // 1GB data
  int myRank, nRanks, localRank = 0;

  // initializing MPI
  MPI_CHECK(MPI_Init(&argc, &argv));
  MPI_CHECK(MPI_Comm_rank(MPI_COMM_WORLD, &myRank));
  MPI_CHECK(MPI_Comm_size(MPI_COMM_WORLD, &nRanks));

  // calculating localRank based on hostname which is used in selecting a GPU
  uint64_t hostHashs[nRanks];
  char hostname[1024];
  getHostName(hostname, 1024);
  hostHashs[myRank] = getHostHash(hostname);
  MPI_CHECK(MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, hostHashs,
			  sizeof(uint64_t), MPI_BYTE, MPI_COMM_WORLD));
  for (int p = 0; p < nRanks; p++) {
    if (p == myRank) {
      break;
    }
    if (hostHashs[p] == hostHashs[myRank]) {
      localRank++;
    }
  }

  ncclUniqueId id[2];
  ncclComm_t comm[2];
  float *sendbuff32, *recvbuff32;
  half  *sendbuff16, *recvbuff16;
  hipStream_t stream[2];

  // get NCCL unique ID at rank 0 and broadcast it to all others
  if (myRank == 0) {
    ncclGetUniqueId(&id[0]);
    ncclGetUniqueId(&id[1]);
  }
  MPI_CHECK(MPI_Bcast((void *)&id[0], sizeof(id[0]), MPI_BYTE, 0, MPI_COMM_WORLD));
  MPI_CHECK(MPI_Bcast((void *)&id[1], sizeof(id[1]), MPI_BYTE, 0, MPI_COMM_WORLD));

  // picking a GPU based on localRank, allocate device buffers
  std::cout << "Picking Device: " << localRank << " for MPI Rank: " << myRank
            << "/" << nRanks << std::endl;
  CUDA_CHECK(hipSetDevice(localRank));

  CUDA_CHECK(hipMalloc(&sendbuff32, size * sizeof(float)));
  CUDA_CHECK(hipMalloc(&recvbuff32, size * sizeof(float)));

  CUDA_CHECK(hipMalloc(&sendbuff16, size * sizeof(half)));
  CUDA_CHECK(hipMalloc(&recvbuff16, size * sizeof(half)));

  CUDA_CHECK(hipStreamCreate(&stream[0]));
  CUDA_CHECK(hipStreamCreate(&stream[1]));

  // initializing NCCL
  std::cout << "Init NCCL for Rank: " << myRank << "/" << nRanks << std::endl;
  NCCL_CHECK(ncclCommInitRank(&comm[0], nRanks, id[0], myRank));
  NCCL_CHECK(ncclCommInitRank(&comm[1], nRanks, id[1], myRank));

  float* tmpCpu32 = (float*)malloc(10 * sizeof(float));
  half*  tmpCpu16 = (half*)malloc(10 * sizeof(half));

  // communicating using NCCL
  for (int i = 0; i < 1; ++i) {
    CUDA_CHECK(hipMemset(sendbuff32, 1.1111 * (i + 1), size * sizeof(float)));
    CUDA_CHECK(hipMemset(sendbuff16, 0.2222 * (i + 1), size * sizeof(half)));

    // std::cout << std::endl << "Communication Test [" << i << "]" << std::endl;
    /*
    if (myRank == 0) {
      hipMemcpy(tmpCpu32, sendbuff32, 10 * sizeof(float), hipMemcpyDeviceToHost);
      for (int j = 1; j < 5; ++j) {
	std::cout << "float32 BEFORE ALLREDUCE MPI RANK " << myRank << ", INDEX "
		  << j << ": " << static_cast<float>(*(tmpCpu32 + j)) << std::endl;
      }
    }
    */

    std::thread reduceThread0;
    std::thread reduceThread1;

    if (myRank == 0) {
      reduceThread0 = std::thread(&delayRunAllReduce, myRank, 1, size, sendbuff32, recvbuff32, comm[1], stream[1]);
      reduceThread1 = std::thread(&delayRunAllReduce, myRank, 0, size, sendbuff16, recvbuff16, comm[0], stream[0]);
      runAllReduce(myRank, 0, size, sendbuff16, recvbuff16, comm[0], stream[0]);
      runAllReduce(myRank, 1, size, sendbuff32, recvbuff32, comm[1], stream[1]);
    }
    if (myRank == 1) {
      reduceThread0 = std::thread(&delayRunAllReduce, myRank, 0, size, sendbuff16, recvbuff16, comm[0], stream[0]);
      reduceThread1 = std::thread(&delayRunAllReduce, myRank, 1, size, sendbuff32, recvbuff32, comm[1], stream[1]);
      runAllReduce(myRank, 1, size, sendbuff32, recvbuff32, comm[1], stream[1]);
      runAllReduce(myRank, 0, size, sendbuff16, recvbuff16, comm[0], stream[0]);
    }

    reduceThread0.join();
    reduceThread1.join();

    /*
    if (myRank == 0) {
      hipMemcpy(tmpCpu32, recvbuff32, 10 * sizeof(float), hipMemcpyDeviceToHost);
      for (int j = 1; j < 5; ++j) {
	std::cout << "float32 AFTER ALLREDUCE MPI RANK " << myRank << ", INDEX "
		  << j << ": " << static_cast<float>(*(tmpCpu32 + j)) << std::endl;
      }
    }
    */
  }

  // free device buffers
  CUDA_CHECK(hipFree(sendbuff32));
  CUDA_CHECK(hipFree(recvbuff32));

  CUDA_CHECK(hipFree(sendbuff16));
  CUDA_CHECK(hipFree(recvbuff16));

  free(tmpCpu32);
  free(tmpCpu16);

  // finalizing NCCL
  ncclCommDestroy(comm[0]);
  ncclCommDestroy(comm[1]);

  // finalizing MPI
  MPI_CHECK(MPI_Finalize());

  std::cout << "[MPI Rank " << myRank << "] Success." << std::endl;
  return 0;
}
