#include <stdio.h>
#include "hip/hip_runtime.h"
#include "nccl.h"
#include "mpi.h"
#include <unistd.h>
#include <stdint.h>
#include <iostream>

#define MPI_CHECK(cmd) do {                      \
  int e = cmd;                                   \
  if (e != MPI_SUCCESS) {                        \
    printf("Failed: MPI error %s:%d '%d'\n",     \
     __FILE__,__LINE__, e);                      \
    exit(EXIT_FAILURE);                          \
  }                                              \
} while(0)

#define CUDA_CHECK(cmd) do {                     \
  hipError_t e = cmd;                           \
  if (e != hipSuccess) {                        \
    printf("Failed: CUDA error %s:%d '%s'\n",    \
     __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                          \
  }                                              \
} while(0)

#define NCCL_CHECK(cmd) do {                     \
  ncclResult_t r = cmd;                          \
  if (r != ncclSuccess) {                        \
    printf("Failed, NCCL error %s:%d '%s'\n",    \
     __FILE__,__LINE__,ncclGetErrorString(r));   \
    exit(EXIT_FAILURE);                          \
  }                                              \
} while(0)

static uint64_t getHostHash(const char *string) {
  // based on DJB2, result = result * 33 + char
  uint64_t result = 5381;
  for (int c = 0; string[c] != '\0'; c++) {
    result = ((result << 5) + result) + string[c];
  }
  return result;
}

static void getHostName(char *hostname, int maxlen) {
  gethostname(hostname, maxlen);
  for (int i = 0; i < maxlen; i++) {
    if (hostname[i] == '.') {
      hostname[i] = '\0';
      return;
    }
  }
}

int main(int argc, char *argv[]) {

  int size = 256 * 1024 * 1024; // 1GB data
  int myRank, nRanks, localRank = 0;

  // initializing MPI
  MPI_CHECK(MPI_Init(&argc, &argv));
  MPI_CHECK(MPI_Comm_rank(MPI_COMM_WORLD, &myRank));
  MPI_CHECK(MPI_Comm_size(MPI_COMM_WORLD, &nRanks));

  // calculating localRank based on hostname which is used in selecting a GPU
  uint64_t hostHashs[nRanks];
  char hostname[1024];
  getHostName(hostname, 1024);
  hostHashs[myRank] = getHostHash(hostname);
  MPI_CHECK(
          MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, hostHashs,
                        sizeof(uint64_t), MPI_BYTE, MPI_COMM_WORLD));
  for (int p = 0; p < nRanks; p++) {
    if (p == myRank) {
      break;
    }
    if (hostHashs[p] == hostHashs[myRank]) {
      localRank++;
    }
  }

  ncclUniqueId id;
  ncclComm_t comm;
  float *sendbuff, *recvbuff;
  hipStream_t s;

  // get NCCL unique ID at rank 0 and broadcast it to all others
  if (myRank == 0) {
    ncclGetUniqueId(&id);
  }
  MPI_CHECK(MPI_Bcast((void *) &id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD));

  // picking a GPU based on localRank, allocate device buffers
  std::cout << "Picking Device: " << localRank << " for MPI Rank: " << myRank
            << "/" << nRanks << std::endl;
  CUDA_CHECK(hipSetDevice(localRank));
  CUDA_CHECK(hipMalloc(&sendbuff, size * sizeof(float)));
  CUDA_CHECK(hipMalloc(&recvbuff, size * sizeof(float)));
  CUDA_CHECK(hipStreamCreate(&s));

  // initializing NCCL
  std::cout << "Init NCCL for Rank: " << myRank << "/" << nRanks << std::endl;
  NCCL_CHECK(ncclCommInitRank(&comm, nRanks, id, myRank));

  // communicating using NCCL
  for (int i = 0; i < 5; ++i) {
    std::cout << "Communicating [" << i << "]" << std::endl;
    NCCL_CHECK(
            ncclAllReduce((const void *) sendbuff, (void *) recvbuff, size,
                          ncclFloat, ncclSum, comm, s));

    // completing NCCL operation by synchronizing on the CUDA stream
    CUDA_CHECK(hipStreamSynchronize(s));
  }

  // free device buffers
  CUDA_CHECK(hipFree(sendbuff));
  CUDA_CHECK(hipFree(recvbuff));

  // finalizing NCCL
  NCCL_CHECK(ncclCommDestroy(comm));

  // finalizing MPI
  MPI_CHECK(MPI_Finalize());

  std::cout << "[MPI Rank " << myRank << "] Success." << std::endl;
  return 0;
}
